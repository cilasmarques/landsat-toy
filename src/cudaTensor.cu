#include <iostream>
#include <hiptensor.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <vector>
#include <assert.h>

using namespace std::chrono;

// Define the matrix size
#define MATRIX_HEIGTH 32768
#define MATRIX_WIDTH 32768

int main()
{
  system_clock::time_point begin, end;
  int64_t initial_time, final_time, general_time;

  // Create a matrix
  float *matrixA = new float[MATRIX_HEIGTH * MATRIX_WIDTH];
  float *matrixB = new float[MATRIX_HEIGTH * MATRIX_WIDTH];
  std::cout << "Matrix size: " << MATRIX_HEIGTH << " x " << MATRIX_WIDTH << std::endl;

  // Initialize the matrix
  for (int i = 0; i < MATRIX_HEIGTH * MATRIX_WIDTH; i++)
  {
    matrixA[i] = 2.0f;
    matrixB[i] = 2.0f;
  }

  // Define the handle
  hiptensorHandle_t handle;
  hiptensorCreate(&handle);

  // Create the tensor descriptors
  int quantidades_de_dimensoes = 2;
  int64_t tamanho_de_cada_dimensao[] = {MATRIX_HEIGTH, MATRIX_WIDTH};
  hiptensorTensorDescriptor_t descA, descB, descC;
  hiptensorStatus_t status;
  status = hiptensorCreateTensorDescriptor(handle, &descA, quantidades_de_dimensoes, tamanho_de_cada_dimensao, NULL, HIPTENSOR_R_32F, HIPTENSOR_OP_IDENTITY);
  status = hiptensorCreateTensorDescriptor(handle, &descB, quantidades_de_dimensoes, tamanho_de_cada_dimensao, NULL, HIPTENSOR_R_32F, HIPTENSOR_OP_IDENTITY);

  // Allocate the memory on the device
  float *d_matrixA, *d_matrixB, *d_matrixC;
  hipMalloc(&d_matrixA, MATRIX_HEIGTH * MATRIX_WIDTH * sizeof(float));
  hipMalloc(&d_matrixB, MATRIX_HEIGTH * MATRIX_WIDTH * sizeof(float));
  hipMalloc(&d_matrixC, MATRIX_HEIGTH * MATRIX_WIDTH * sizeof(float));

  // Copy the data to the device
  hipMemcpy(d_matrixA, matrixA, MATRIX_HEIGTH * MATRIX_WIDTH * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_matrixB, matrixB, MATRIX_HEIGTH * MATRIX_WIDTH * sizeof(float), hipMemcpyHostToDevice);

  // Create the tensor operations
  const int32_t eixosA[] = {MATRIX_HEIGTH, MATRIX_WIDTH};
  const int32_t eixosB[] = {MATRIX_HEIGTH, MATRIX_WIDTH};
  const int32_t eixosC[] = {MATRIX_HEIGTH, MATRIX_WIDTH};

  hiptensorOperationDescriptor_t desc;
  hiptensorComputeDescriptor_t descCompute = HIPTENSOR_COMPUTE_DESC_32F;
  hiptensorCreateContraction(
      handle,
      &desc,
      descA, eixosA, HIPTENSOR_OP_IDENTITY, // Indica que não há transformação adicional em A antes da operação binária.
      descB, eixosB, HIPTENSOR_OP_IDENTITY, // Indica que não há transformação adicional em B antes da operação binária.
      NULL, NULL, HIPTENSOR_OP_IDENTITY,    // Indica que não há transformação adicional em C antes da operação binária.
      descC, eixosC, descCompute);

  // Ensure that the scalar type is correct.
  hiptensorDataType_t scalarType;
  hiptensorOperationDescriptorGetAttribute(handle,
                                          desc,
                                          HIPTENSOR_OPERATION_DESCRIPTOR_SCALAR_TYPE,
                                          (void *)&scalarType,
                                          sizeof(scalarType));

  assert(scalarType == HIPTENSOR_R_32F);

  // Define scaling factors
  typedef float floatTypeCompute;
  floatTypeCompute alpha = (floatTypeCompute)1.1f;
  floatTypeCompute beta = (floatTypeCompute)0.f;

  // Set the algorithm to use
  const hiptensorAlgo_t algo = HIPTENSOR_ALGO_DEFAULT;
  hiptensorPlanPreference_t planPref;
  hiptensorCreatePlanPreference(handle, &planPref, algo, HIPTENSOR_JIT_MODE_NONE);

  // Estimate the workspace size
  uint64_t workspaceSizeEstimate = 0;
  const hiptensorWorksizePreference_t workspacePref = HIPTENSOR_WORKSPACE_DEFAULT;
  hiptensorEstimateWorkspaceSize(handle, desc, planPref, workspacePref, &workspaceSizeEstimate);

  // Create the plan
  hiptensorPlan_t plan;
  hiptensorCreatePlan(handle, &plan, desc, planPref, workspaceSizeEstimate);

  // Optional: Query information about the created plan
  uint64_t actualWorkspaceSize = 0;
  hiptensorPlanGetAttribute(handle,plan,HIPTENSOR_PLAN_REQUIRED_WORKSPACE,&actualWorkspaceSize,sizeof(actualWorkspaceSize));
  assert(actualWorkspaceSize <= workspaceSizeEstimate);

  void *work = nullptr;
  if (actualWorkspaceSize > 0)
  {
    hipMalloc(&work, actualWorkspaceSize);
    assert(uintptr_t(work) % 128 == 0); // workspace must be aligned to 128 byte-boundary
  }

  hipStream_t stream;
  hipStreamCreate(&stream);

  // Execute the operation
  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  hiptensorContract(handle, plan, (void *)&alpha, d_matrixA, d_matrixB, (void *)&beta, d_matrixC, d_matrixC, work, actualWorkspaceSize, stream);

  end = system_clock::now();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  general_time = duration_cast<nanoseconds>(end.time_since_epoch() - begin.time_since_epoch()).count();
  std::cout << "CUBLAS CORE - TOTAL TIME (ns): " << general_time << std::endl;
  // ======== RUN CUBLAS TENSOR =========

  // Copy the result back to the host
  float *matrixC = new float[MATRIX_HEIGTH * MATRIX_WIDTH];
  hipMemcpy(matrixC, d_matrixC, MATRIX_HEIGTH * MATRIX_WIDTH * sizeof(float), hipMemcpyDeviceToHost);

  // Print the result
  // std::cout << "Matrix C: " << std::endl;
  // for (int i = 0; i < MATRIX_HEIGTH; i++)
  // {
  //   for (int j = 0; j < MATRIX_WIDTH; j++)
  //   {
  //     std::cout << matrixC[i * MATRIX_WIDTH + j] << " ";
  //   }
  //   std::cout << std::endl;
  // }

  // Free the memory
  delete[] matrixA;
  delete[] matrixB;
  delete[] matrixC;
  hipFree(d_matrixA);
  hipFree(d_matrixB);
  hipFree(d_matrixC);

  return 0;
}