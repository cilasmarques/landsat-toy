/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are
 * met:
 *  - Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  - Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  - Neither the name(s) of the copyright holder(s) nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
 * A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
 * HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
 * SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
 * LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <iostream>

#include <unordered_map>
#include <vector>
#include <chrono>

#include <hip/hip_runtime.h>
#include <hiptensor.h>

using namespace std::chrono;

#define HANDLE_ERROR(x)                                   \
  {                                                       \
    const auto err = x;                                   \
    if (err != HIPTENSOR_STATUS_SUCCESS)                   \
    {                                                     \
      printf("Error: %s\n", hiptensorGetErrorString(err)); \
      exit(-1);                                           \
    }                                                     \
  };

#define HANDLE_CUDA_ERROR(x)                          \
  {                                                   \
    const auto err = x;                               \
    if (err != hipSuccess)                           \
    {                                                 \
      printf("Error: %s\n", hipGetErrorString(err)); \
      exit(-1);                                       \
    }                                                 \
  };

struct GPUTimer
{
  GPUTimer()
  {
    hipEventCreate(&start_);
    hipEventCreate(&stop_);
    hipEventRecord(start_, 0);
  }

  ~GPUTimer()
  {
    hipEventDestroy(start_);
    hipEventDestroy(stop_);
  }

  void start()
  {
    hipEventRecord(start_, 0);
  }

  float seconds()
  {
    hipEventRecord(stop_, 0);
    hipEventSynchronize(stop_);
    float time;
    hipEventElapsedTime(&time, start_, stop_);
    return time * 1e-3;
  }

private:
  hipEvent_t start_, stop_;
};

int main()
{
  system_clock::time_point begin, end;
  int64_t general_time;

  typedef float floatTypeA;
  typedef float floatTypeB;
  typedef float floatTypeC;

  /**********************
   * Computing: C_{m,n} = alpha * A_{m,n} B_{m,n} + beta * C_{m,n}
   **********************/

  // Define o tamanho de cada dimensão
  std::vector<int64_t> extentA = {10000, 10000};
  std::vector<int64_t> extentB = {10000, 10000};
  std::vector<int64_t> extentC = {10000, 10000};

  // Define a quantidade de dimensões dos tensores
  int nmodeA = 2;
  int nmodeB = 2;
  int nmodeC = 2;

  /**********************
   * Allocating data
   **********************/
  size_t elementsA = extentA[0] * extentA[1];
  size_t elementsB = extentB[0] * extentB[1];
  size_t elementsC = extentC[0] * extentC[1];

  size_t sizeA = sizeof(floatTypeA) * elementsA;
  size_t sizeB = sizeof(floatTypeB) * elementsB;
  size_t sizeC = sizeof(floatTypeC) * elementsC;
  printf("Total memory: %.2f GiB\n", (sizeA + sizeB + sizeC) / 1024. / 1024. / 1024);

  void *A_d, *B_d, *C_d;
  HANDLE_CUDA_ERROR(hipMalloc((void **)&A_d, sizeA));
  HANDLE_CUDA_ERROR(hipMalloc((void **)&B_d, sizeB));
  HANDLE_CUDA_ERROR(hipMalloc((void **)&C_d, sizeC));

  floatTypeA *A = (floatTypeA *)malloc(sizeof(floatTypeA) * elementsA);
  floatTypeB *B = (floatTypeB *)malloc(sizeof(floatTypeB) * elementsB);
  floatTypeC *C = (floatTypeC *)malloc(sizeof(floatTypeC) * elementsC);

  if (A == NULL || B == NULL || C == NULL)
  {
    printf("Error: Host allocation of A or C.\n");
    return -1;
  }

  /*******************
   * Initialize data
   *******************/

  for (int64_t i = 0; i < elementsA; i++)
    A[i] = 4;
  for (int64_t i = 0; i < elementsB; i++)
    B[i] = 2;
  for (int64_t i = 0; i < elementsC; i++)
    C[i] = 0;

  // print A
  printf("A size: %ld\n", sizeA);

  HANDLE_CUDA_ERROR(hipMemcpy(A_d, A, sizeA, hipMemcpyHostToDevice));
  HANDLE_CUDA_ERROR(hipMemcpy(B_d, B, sizeB, hipMemcpyHostToDevice));
  HANDLE_CUDA_ERROR(hipMemcpy(C_d, C, sizeC, hipMemcpyHostToDevice));

  const uint32_t kAlignment = 128; // Alignment of the global-memory device pointers (bytes)
  assert(uintptr_t(A_d) % kAlignment == 0);
  assert(uintptr_t(B_d) % kAlignment == 0);
  assert(uintptr_t(C_d) % kAlignment == 0);

  /*************************
   * cuTENSOR
   *************************/

  hiptensorHandle_t handle;
  HANDLE_ERROR(hiptensorCreate(&handle));

  /**********************
   * Create Tensor Descriptors
   **********************/

  hiptensorTensorDescriptor_t descA;
  HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle,
                                              &descA,
                                              nmodeA,
                                              extentA.data(),
                                              NULL, /*stride*/
                                              HIPTENSOR_R_32F, kAlignment));

  hiptensorTensorDescriptor_t descB;
  HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle,
                                              &descB,
                                              nmodeB,
                                              extentB.data(),
                                              NULL, /*stride*/
                                              HIPTENSOR_R_32F, kAlignment));

  hiptensorTensorDescriptor_t descC;
  HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle,
                                              &descC,
                                              nmodeC,
                                              extentC.data(),
                                              NULL, /*stride*/
                                              HIPTENSOR_R_32F, kAlignment));

  /*******************************
   * Create Contraction Descriptor
   *******************************/
  std::vector<int> modeC{'m', 'n'};
  std::vector<int> modeA{'m', 'n'};
  std::vector<int> modeB{'m', 'n'};

  hiptensorOperationDescriptor_t desc;
  const hiptensorComputeDescriptor_t descCompute = HIPTENSOR_COMPUTE_DESC_32F;
  HANDLE_ERROR(hiptensorCreateContraction(handle,
                                         &desc,
                                         descA, modeA.data(), /* unary operator A*/ HIPTENSOR_OP_IDENTITY,
                                         descB, modeB.data(), /* unary operator B*/ HIPTENSOR_OP_IDENTITY,
                                         descC, modeC.data(), /* unary operator C*/ HIPTENSOR_OP_IDENTITY,
                                         descC, modeC.data(),
                                         descCompute));

  /*****************************
   * Optional (but recommended): ensure that the scalar type is correct.
   *****************************/

  hiptensorDataType_t scalarType;
  HANDLE_ERROR(hiptensorOperationDescriptorGetAttribute(handle,
                                                       desc,
                                                       HIPTENSOR_OPERATION_DESCRIPTOR_SCALAR_TYPE,
                                                       (void *)&scalarType,
                                                       sizeof(scalarType)));

  assert(scalarType == HIPTENSOR_R_32F);
  typedef float floatTypeCompute;
  floatTypeCompute alpha = (floatTypeCompute)1;
  floatTypeCompute beta = (floatTypeCompute)0;

  /**************************
   * Set the algorithm to use
   ***************************/

  const hiptensorAlgo_t algo = HIPTENSOR_ALGO_DEFAULT;

  hiptensorPlanPreference_t planPref;
  HANDLE_ERROR(hiptensorCreatePlanPreference(
      handle,
      &planPref,
      algo,
      HIPTENSOR_JIT_MODE_NONE));

  /**********************
   * Query workspace estimate
   **********************/

  uint64_t workspaceSizeEstimate = 0;
  const hiptensorWorksizePreference_t workspacePref = HIPTENSOR_WORKSPACE_DEFAULT;
  HANDLE_ERROR(hiptensorEstimateWorkspaceSize(handle,
                                             desc,
                                             planPref,
                                             workspacePref,
                                             &workspaceSizeEstimate));

  /**************************
   * Create Contraction Plan
   **************************/

  hiptensorPlan_t plan;
  HANDLE_ERROR(hiptensorCreatePlan(handle,
                                  &plan,
                                  desc,
                                  planPref,
                                  workspaceSizeEstimate));

  /**************************
   * Optional: Query information about the created plan
   **************************/

  // query actually used workspace
  uint64_t actualWorkspaceSize = 0;
  HANDLE_ERROR(hiptensorPlanGetAttribute(handle,
                                        plan,
                                        HIPTENSOR_PLAN_REQUIRED_WORKSPACE,
                                        &actualWorkspaceSize,
                                        sizeof(actualWorkspaceSize)));

  // At this point the user knows exactly how much memory is need by the operation and
  // only the smaller actual workspace needs to be allocated
  assert(actualWorkspaceSize <= workspaceSizeEstimate);

  void *work = nullptr;
  if (actualWorkspaceSize > 0)
  {
    HANDLE_CUDA_ERROR(hipMalloc(&work, actualWorkspaceSize));
    assert(uintptr_t(work) % 128 == 0); // workspace must be aligned to 128 byte-boundary
  }

  /**********************
   * Run
   **********************/

  hipStream_t stream;
  HANDLE_CUDA_ERROR(hipStreamCreate(&stream));

  begin = system_clock::now();

  HANDLE_ERROR(hiptensorContract(handle,
                                plan,
                                (void *)&alpha, A_d, B_d,
                                (void *)&beta, C_d, C_d,
                                work, actualWorkspaceSize, stream));

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end.time_since_epoch() - begin.time_since_epoch()).count();
  std::cout << "CUTENSOR CORE - TOTAL TIME (ns): " << general_time << std::endl;

  // // print the result in a aux variable
  // floatTypeC *C_aux = (floatTypeC*) malloc(sizeof(floatTypeC) * elementsC);

  // // Copy the result back to the host
  // HANDLE_CUDA_ERROR(hipMemcpy(C_aux, C_d, sizeC, hipMemcpyDeviceToHost));

  // // Print the result
  // std::cout << "Matrix C: " << std::endl;
  // for (int i = 0; i < extentC[0]; i++)
  // {
  //     for (int j = 0; j < extentC[1]; j++)
  //     {
  //         std::cout << C_aux[i * extentC[1] + j] << " ";
  //     }
  //     std::cout << std::endl;
  // }

  /*************************/

  HANDLE_ERROR(hiptensorDestroy(handle));
  HANDLE_ERROR(hiptensorDestroyPlan(plan));
  HANDLE_ERROR(hiptensorDestroyOperationDescriptor(desc));
  HANDLE_ERROR(hiptensorDestroyTensorDescriptor(descA));
  HANDLE_ERROR(hiptensorDestroyTensorDescriptor(descB));
  HANDLE_ERROR(hiptensorDestroyTensorDescriptor(descC));
  HANDLE_CUDA_ERROR(hipStreamDestroy(stream));

  if (A)
    free(A);
  if (B)
    free(B);
  if (C)
    free(C);
  if (A_d)
    hipFree(A_d);
  if (B_d)
    hipFree(B_d);
  if (C_d)
    hipFree(C_d);
  if (work)
    hipFree(work);

  return 0;
}