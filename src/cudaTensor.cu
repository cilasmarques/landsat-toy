/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are
 * met:
 *  - Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  - Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  - Neither the name(s) of the copyright holder(s) nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
 * A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
 * HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
 * SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
 * LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <iostream>

#include <unordered_map>
#include <vector>
#include <chrono>

#include <hip/hip_runtime.h>
#include <hiptensor.h>

using namespace std::chrono;

#define MATRIX_HEIGTH 128*64
#define MATRIX_WIDTH 128*64

#define HANDLE_ERROR(x)                                   \
  {                                                       \
    const auto err = x;                                   \
    if (err != HIPTENSOR_STATUS_SUCCESS)                   \
    {                                                     \
      printf("Error: %s\n", hiptensorGetErrorString(err)); \
      exit(-1);                                           \
    }                                                     \
  };

#define HANDLE_CUDA_ERROR(x)                          \
  {                                                   \
    const auto err = x;                               \
    if (err != hipSuccess)                           \
    {                                                 \
      printf("Error: %s\n", hipGetErrorString(err)); \
      exit(-1);                                       \
    }                                                 \
  };

struct GPUTimer
{
  GPUTimer()
  {
    hipEventCreate(&start_);
    hipEventCreate(&stop_);
    hipEventRecord(start_, 0);
  }

  ~GPUTimer()
  {
    hipEventDestroy(start_);
    hipEventDestroy(stop_);
  }

  void start()
  {
    hipEventRecord(start_, 0);
  }

  float seconds()
  {
    hipEventRecord(stop_, 0);
    hipEventSynchronize(stop_);
    float time;
    hipEventElapsedTime(&time, start_, stop_);
    return time * 1e-3;
  }

private:
  hipEvent_t start_, stop_;
};

/**********************
 * Computing: C_{m,n} = alpha * A_{m,n} B_{m,n} + beta * C_{m,n}
 **********************/
int main()
{
  system_clock::time_point begin, end;
  int64_t general_time;

  // Define os eixos
  int tamanhoEixoA = 2;
  int tamanhoEixoB = 2;
  int tamanhoEixoC = 2;
  std::vector<int> indicesEixoA{'m', 'n'};
  // std::vector<int> indicesEixoB{'m', 'n'}; // Produto Escalar
  std::vector<int> indicesEixoB{'n', 'j'}; // Produto de hadamard
  std::vector<int> indicesEixoC{'m', 'n'};
  std::vector<int64_t> dimensoesEixoA = {MATRIX_HEIGTH, MATRIX_WIDTH};
  std::vector<int64_t> dimensoesEixoB = {MATRIX_HEIGTH, MATRIX_WIDTH};
  std::vector<int64_t> dimensoesEixoC = {MATRIX_HEIGTH, MATRIX_WIDTH};

  // Aloca memória
  void *A_d, *B_d, *C_d;
  HANDLE_CUDA_ERROR(hipMalloc((void **)&A_d, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH));
  HANDLE_CUDA_ERROR(hipMalloc((void **)&B_d, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH));
  HANDLE_CUDA_ERROR(hipMalloc((void **)&C_d, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH));

  float *A = (float *)malloc(sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH);
  float *B = (float *)malloc(sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH);
  float *C = (float *)malloc(sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH);

  // Alignment of the global-memory device pointers (bytes)
  const uint32_t kAlignment = 128;
  assert(uintptr_t(A_d) % kAlignment == 0);
  assert(uintptr_t(B_d) % kAlignment == 0);
  assert(uintptr_t(C_d) % kAlignment == 0);

  // Inicializa a matrix
  for (int i = 0; i < MATRIX_HEIGTH * MATRIX_WIDTH; i++)
  {
    A[i] = 4.0f;
    B[i] = 2.0f;
    C[i] = 0.0f;
  }

  // Copia os dados para o dispositivo
  HANDLE_CUDA_ERROR(hipMemcpy(A_d, A, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH, hipMemcpyHostToDevice));
  HANDLE_CUDA_ERROR(hipMemcpy(B_d, B, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH, hipMemcpyHostToDevice));
  HANDLE_CUDA_ERROR(hipMemcpy(C_d, C, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH, hipMemcpyHostToDevice));

  // Define o handle cutensor
  hiptensorHandle_t handle;
  HANDLE_ERROR(hiptensorCreate(&handle));

  // Define os descriptors
  hiptensorTensorDescriptor_t descA;
  hiptensorTensorDescriptor_t descB;
  hiptensorTensorDescriptor_t descC;
  HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle, &descA, tamanhoEixoA, dimensoesEixoA.data(), NULL, HIPTENSOR_R_32F, kAlignment));
  HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle, &descB, tamanhoEixoB, dimensoesEixoB.data(), NULL, HIPTENSOR_R_32F, kAlignment));
  HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle, &descC, tamanhoEixoC, dimensoesEixoC.data(), NULL, HIPTENSOR_R_32F, kAlignment));

  // Create Contraction Descriptor
  hiptensorOperationDescriptor_t desc;
  const hiptensorComputeDescriptor_t descCompute = HIPTENSOR_COMPUTE_DESC_32F;
  HANDLE_ERROR(hiptensorCreateContraction(handle,
                                         &desc,
                                         descA, indicesEixoA.data(), /* unary operator A*/ HIPTENSOR_OP_IDENTITY,
                                         descB, indicesEixoB.data(), /* unary operator B*/ HIPTENSOR_OP_IDENTITY,
                                         descC, indicesEixoC.data(), /* unary operator C*/ HIPTENSOR_OP_IDENTITY,
                                         descC, indicesEixoC.data(),
                                         descCompute));

  // Optional (but recommended): ensure that the scalar type is correct.
  hiptensorDataType_t scalarType;
  HANDLE_ERROR(hiptensorOperationDescriptorGetAttribute(handle, desc, HIPTENSOR_OPERATION_DESCRIPTOR_SCALAR_TYPE, (void *)&scalarType, sizeof(scalarType)));

  assert(scalarType == HIPTENSOR_R_32F);
  typedef float floatTypeCompute;
  floatTypeCompute alpha = (floatTypeCompute)1;
  floatTypeCompute beta = (floatTypeCompute)0;

  // Set the algorithm to use
  const hiptensorAlgo_t algo = HIPTENSOR_ALGO_DEFAULT;
  hiptensorPlanPreference_t planPref;
  HANDLE_ERROR(hiptensorCreatePlanPreference(handle, &planPref, algo, HIPTENSOR_JIT_MODE_NONE));

  // Query workspace estimate
  uint64_t workspaceSizeEstimate = 0;
  const hiptensorWorksizePreference_t workspacePref = HIPTENSOR_WORKSPACE_DEFAULT;
  HANDLE_ERROR(hiptensorEstimateWorkspaceSize(handle, desc, planPref, workspacePref, &workspaceSizeEstimate));

  // Create Contraction Plan
  hiptensorPlan_t plan;
  HANDLE_ERROR(hiptensorCreatePlan(handle, &plan, desc, planPref, workspaceSizeEstimate));

  // Optional: query actually used workspace
  uint64_t actualWorkspaceSize = 0;
  HANDLE_ERROR(hiptensorPlanGetAttribute(handle, plan, HIPTENSOR_PLAN_REQUIRED_WORKSPACE, &actualWorkspaceSize, sizeof(actualWorkspaceSize)));
  assert(actualWorkspaceSize <= workspaceSizeEstimate);

  // Define the workspace
  void *work = nullptr;
  if (actualWorkspaceSize > 0)
  {
    HANDLE_CUDA_ERROR(hipMalloc(&work, actualWorkspaceSize));
    assert(uintptr_t(work) % 128 == 0); // workspace must be aligned to 128 byte-boundary
  }

  // Execute
  hipStream_t stream;
  HANDLE_CUDA_ERROR(hipStreamCreate(&stream));

  begin = system_clock::now();

  HANDLE_ERROR(hiptensorContract(handle, plan, (void *)&alpha, A_d, B_d, (void *)&beta, C_d, C_d, work, actualWorkspaceSize, stream));

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end.time_since_epoch() - begin.time_since_epoch()).count();
  std::cout << "CUTENSOR," << MATRIX_HEIGTH << " x " << MATRIX_WIDTH << ", " << general_time << std::endl;

  // Print the result
  float *C_aux = (float *)malloc(sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH);
  HANDLE_CUDA_ERROR(hipMemcpy(C_aux, C_d, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH, hipMemcpyDeviceToHost));

  // Print the result
  // std::cout << "Matrix C: " << std::endl;
  // for (int i = 0; i < dimensoesEixoC[0]; i++)
  // {
  //   for (int j = 0; j < dimensoesEixoC[1]; j++)
  //   {
  //     std::cout << C_aux[i * dimensoesEixoC[1] + j] << " ";
  //   }
  //   std::cout << std::endl;
  // }

  HANDLE_ERROR(hiptensorDestroy(handle));
  HANDLE_ERROR(hiptensorDestroyPlan(plan));
  HANDLE_ERROR(hiptensorDestroyOperationDescriptor(desc));
  HANDLE_ERROR(hiptensorDestroyTensorDescriptor(descA));
  HANDLE_ERROR(hiptensorDestroyTensorDescriptor(descB));
  HANDLE_ERROR(hiptensorDestroyTensorDescriptor(descC));
  HANDLE_CUDA_ERROR(hipStreamDestroy(stream));

  if (A)
    free(A);
  if (B)
    free(B);
  if (C)
    free(C);
  if (A_d)
    hipFree(A_d);
  if (B_d)
    hipFree(B_d);
  if (C_d)
    hipFree(C_d);
  if (work)
    hipFree(work);

  return 0;
}