#include <iostream>
#include <hiptensor.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <vector>

using namespace std::chrono;

// Define the matrix size
#define MATRIX_HEIGTH 32768
#define MATRIX_WIDTH 32768

void cublasTensorExecution(hipblasHandle_t cubaslHandle, float *d_matrixA, float *d_matrixB, float *d_matrixC)
{
  // Perform matrix sum: C = A / B
  float alpha = 1.0f;
  float beta = 1.0f;
  hipblasStatus_t cublasStat = hipblasSgeam(cubaslHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, MATRIX_HEIGTH, MATRIX_WIDTH, &alpha, d_matrixA, MATRIX_HEIGTH, &beta, d_matrixB, MATRIX_HEIGTH, d_matrixC, MATRIX_HEIGTH);
}

int main()
{
  system_clock::time_point begin, end;
  int64_t initial_time, final_time, general_time;

  // Create a matrix
  float *matrixA = new float[MATRIX_HEIGTH * MATRIX_WIDTH];
  float *matrixB = new float[MATRIX_HEIGTH * MATRIX_WIDTH];
  std::cout << "Matrix size: " << MATRIX_HEIGTH << " x " << MATRIX_WIDTH << std::endl;

  // Initialize the matrix
  for (int i = 0; i < MATRIX_HEIGTH * MATRIX_WIDTH; i++)
  {
    matrixA[i] = 2.0f;
    matrixB[i] = 2.0f;
  }

  // Allocate the memory on the device
  float *d_matrixA;
  float *d_matrixB;
  float *d_matrixC;
  hipMalloc(&d_matrixA, MATRIX_HEIGTH * MATRIX_WIDTH * sizeof(float));
  hipMalloc(&d_matrixB, MATRIX_HEIGTH * MATRIX_WIDTH * sizeof(float));
  hipMalloc(&d_matrixC, MATRIX_HEIGTH * MATRIX_WIDTH * sizeof(float));

  // Copy the data to the device
  hipMemcpy(d_matrixA, matrixA, MATRIX_HEIGTH * MATRIX_WIDTH * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_matrixB, matrixB, MATRIX_HEIGTH * MATRIX_WIDTH * sizeof(float), hipMemcpyHostToDevice);

  // ======== RUN CUBLAS TENSOR =========
  // Define the handle
  hipblasHandle_t cubaslHandle;
  hipblasStatus_t cublasStat = hipblasCreate(&cubaslHandle);

  // Set the math mode to allow cuBLAS to use Tensor Cores:
  cublasStat = hipblasSetMathMode(cubaslHandle, HIPBLAS_TENSOR_OP_MATH);

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  cublasTensorExecution(cubaslHandle, d_matrixA, d_matrixB, d_matrixC);

  end = system_clock::now();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  general_time = duration_cast<nanoseconds>(end.time_since_epoch() - begin.time_since_epoch()).count();
  std::cout << "CUBLAS CORE - TOTAL TIME (ns): " << general_time << std::endl;
  // ======== RUN CUBLAS TENSOR =========

  // Copy the result back to the host
  float *matrixC = new float[MATRIX_HEIGTH * MATRIX_WIDTH];
  hipMemcpy(matrixC, d_matrixC, MATRIX_HEIGTH * MATRIX_WIDTH * sizeof(float), hipMemcpyDeviceToHost);

  // Print the result
  // std::cout << "Matrix C: " << std::endl;
  // for (int i = 0; i < MATRIX_HEIGTH; i++)
  // {
  //   for (int j = 0; j < MATRIX_WIDTH; j++)
  //   {
  //     std::cout << matrixC[i * MATRIX_WIDTH + j] << " ";
  //   }
  //   std::cout << std::endl;
  // }

  // Free the memory
  delete[] matrixA;
  delete[] matrixB;
  delete[] matrixC;
  hipFree(d_matrixA);
  hipFree(d_matrixB);
  hipFree(d_matrixC);

  return 0;
}