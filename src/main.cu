#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <vector>
#include <thread>
#include <tiffio.h>
#include <string>
#include <assert.h>
#include <iostream>

#include "parameters.h"

#include <hip/hip_runtime.h>
#include <hiptensor.h>

using namespace std::chrono;

// Define the matrix size
#define MATRIX_HEIGTH 35
#define MATRIX_WIDTH 35

double *band1 = new double[MATRIX_HEIGTH * MATRIX_WIDTH];
double *band2 = new double[MATRIX_HEIGTH * MATRIX_WIDTH];
double *band3 = new double[MATRIX_HEIGTH * MATRIX_WIDTH];
double *band4 = new double[MATRIX_HEIGTH * MATRIX_WIDTH];
double *band5 = new double[MATRIX_HEIGTH * MATRIX_WIDTH];
double *band6 = new double[MATRIX_HEIGTH * MATRIX_WIDTH];
double *band7 = new double[MATRIX_HEIGTH * MATRIX_WIDTH];
double *band8 = new double[MATRIX_HEIGTH * MATRIX_WIDTH];

double *radiance1 = new double[MATRIX_HEIGTH * MATRIX_WIDTH];
double *radiance2 = new double[MATRIX_HEIGTH * MATRIX_WIDTH];
double *radiance3 = new double[MATRIX_HEIGTH * MATRIX_WIDTH];
double *radiance4 = new double[MATRIX_HEIGTH * MATRIX_WIDTH];
double *radiance5 = new double[MATRIX_HEIGTH * MATRIX_WIDTH];
double *radiance6 = new double[MATRIX_HEIGTH * MATRIX_WIDTH];
double *radiance7 = new double[MATRIX_HEIGTH * MATRIX_WIDTH];
double *radiance8 = new double[MATRIX_HEIGTH * MATRIX_WIDTH];

double *reflectance1 = new double[MATRIX_HEIGTH * MATRIX_WIDTH];
double *reflectance2 = new double[MATRIX_HEIGTH * MATRIX_WIDTH];
double *reflectance3 = new double[MATRIX_HEIGTH * MATRIX_WIDTH];
double *reflectance4 = new double[MATRIX_HEIGTH * MATRIX_WIDTH];
double *reflectance5 = new double[MATRIX_HEIGTH * MATRIX_WIDTH];
double *reflectance6 = new double[MATRIX_HEIGTH * MATRIX_WIDTH];
double *reflectance7 = new double[MATRIX_HEIGTH * MATRIX_WIDTH];
double *reflectance8 = new double[MATRIX_HEIGTH * MATRIX_WIDTH];

int serial(Sensor sensor, MTL mtl)
{
  int64_t general_time;
  system_clock::time_point begin, end;

  // ======== RUN =========
  begin = system_clock::now();

  for (int i = 0; i < MATRIX_HEIGTH * MATRIX_WIDTH; i++)
  {
    radiance1[i] = band1[i] * sensor.parameters[1][sensor.GRESCALE] + sensor.parameters[1][sensor.BRESCALE];
    radiance2[i] = band2[i] * sensor.parameters[2][sensor.GRESCALE] + sensor.parameters[2][sensor.BRESCALE];
    radiance3[i] = band3[i] * sensor.parameters[3][sensor.GRESCALE] + sensor.parameters[3][sensor.BRESCALE];
    radiance4[i] = band4[i] * sensor.parameters[4][sensor.GRESCALE] + sensor.parameters[4][sensor.BRESCALE];
    radiance5[i] = band5[i] * sensor.parameters[5][sensor.GRESCALE] + sensor.parameters[5][sensor.BRESCALE];
    radiance6[i] = band6[i] * sensor.parameters[6][sensor.GRESCALE] + sensor.parameters[6][sensor.BRESCALE];
    radiance7[i] = band7[i] * sensor.parameters[7][sensor.GRESCALE] + sensor.parameters[7][sensor.BRESCALE];
    radiance8[i] = band8[i] * sensor.parameters[8][sensor.GRESCALE] + sensor.parameters[8][sensor.BRESCALE];
  }

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end.time_since_epoch() - begin.time_since_epoch()).count();
  std::cout << "CUDA CORE," << MATRIX_HEIGTH << " x " << MATRIX_WIDTH << ", " << general_time << std::endl;
  // ======== RUN =========

  // ======== RUN =========
  begin = system_clock::now();

  const float sin_sun = sin(mtl.sun_elevation * PI / 180);

  for (int i = 0; i < MATRIX_HEIGTH * MATRIX_WIDTH; i++)
  {
    reflectance1[i] = radiance1[i] / sin_sun;
    reflectance2[i] = radiance2[i] / sin_sun;
    reflectance3[i] = radiance3[i] / sin_sun;
    reflectance4[i] = radiance4[i] / sin_sun;
    reflectance5[i] = radiance5[i] / sin_sun;
    reflectance6[i] = radiance6[i] / sin_sun;
    reflectance7[i] = radiance7[i] / sin_sun;
    reflectance8[i] = radiance8[i] / sin_sun;
  }

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end.time_since_epoch() - begin.time_since_epoch()).count();
  std::cout << "CUDA CORE," << MATRIX_HEIGTH << " x " << MATRIX_WIDTH << ", " << general_time << std::endl;
  // ======== RUN =========
}

void cutensor(std::vector<double *> bands, std::vector<double *> rad_bands, std::vector<double *> ref_bands, Sensor sensor, MTL mtl)
{
  int64_t general_time;
  system_clock::time_point begin, end;

  // Define os eixos
  int tamanhoEixoA = 2;
  int tamanhoEixoB = 2;
  int tamanhoEixoC = 2;
  std::vector<int> indicesEixoA{'m', 'n'};
  std::vector<int> indicesEixoB{'m', 'n'};
  std::vector<int> indicesEixoC{'m', 'n'};
  std::vector<int64_t> dimensoesEixoA = {MATRIX_HEIGTH, MATRIX_WIDTH};
  std::vector<int64_t> dimensoesEixoB = {MATRIX_HEIGTH, MATRIX_WIDTH};
  std::vector<int64_t> dimensoesEixoC = {MATRIX_HEIGTH, MATRIX_WIDTH};

  // Aloca memória
  void *band1_d, *band2_d, *band3_d, *band4_d, *band5_d, *band6_d, *band7_d, *band8_d;
  void *radiance1_d, *radiance2_d, *radiance3_d, *radiance4_d, *radiance5_d, *radiance6_d, *radiance7_d, *radiance8_d;

  HANDLE_CUDA_ERROR(hipMalloc((void **)&band1_d, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH));
  HANDLE_CUDA_ERROR(hipMalloc((void **)&band2_d, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH));
  HANDLE_CUDA_ERROR(hipMalloc((void **)&band3_d, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH));
  HANDLE_CUDA_ERROR(hipMalloc((void **)&band4_d, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH));
  HANDLE_CUDA_ERROR(hipMalloc((void **)&band5_d, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH));
  HANDLE_CUDA_ERROR(hipMalloc((void **)&band6_d, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH));
  HANDLE_CUDA_ERROR(hipMalloc((void **)&band7_d, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH));
  HANDLE_CUDA_ERROR(hipMalloc((void **)&band8_d, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH));

  HANDLE_CUDA_ERROR(hipMalloc((void **)&radiance1_d, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH));
  HANDLE_CUDA_ERROR(hipMalloc((void **)&radiance2_d, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH));
  HANDLE_CUDA_ERROR(hipMalloc((void **)&radiance3_d, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH));
  HANDLE_CUDA_ERROR(hipMalloc((void **)&radiance4_d, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH));
  HANDLE_CUDA_ERROR(hipMalloc((void **)&radiance5_d, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH));
  HANDLE_CUDA_ERROR(hipMalloc((void **)&radiance6_d, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH));
  HANDLE_CUDA_ERROR(hipMalloc((void **)&radiance7_d, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH));
  HANDLE_CUDA_ERROR(hipMalloc((void **)&radiance8_d, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH));

  // Alignment of the global-memory device pointers (bytes)
  const uint32_t kAlignment = 128;
  assert(uintptr_t(band1_d) % kAlignment == 0);
  assert(uintptr_t(band2_d) % kAlignment == 0);
  assert(uintptr_t(band3_d) % kAlignment == 0);
  assert(uintptr_t(band4_d) % kAlignment == 0);
  assert(uintptr_t(band5_d) % kAlignment == 0);
  assert(uintptr_t(band6_d) % kAlignment == 0);
  assert(uintptr_t(band7_d) % kAlignment == 0);
  assert(uintptr_t(band8_d) % kAlignment == 0);

  assert(uintptr_t(radiance1_d) % kAlignment == 0);
  assert(uintptr_t(radiance2_d) % kAlignment == 0);
  assert(uintptr_t(radiance3_d) % kAlignment == 0);
  assert(uintptr_t(radiance4_d) % kAlignment == 0);
  assert(uintptr_t(radiance5_d) % kAlignment == 0);
  assert(uintptr_t(radiance6_d) % kAlignment == 0);
  assert(uintptr_t(radiance7_d) % kAlignment == 0);
  assert(uintptr_t(radiance8_d) % kAlignment == 0);

  // Copia os dados para o dispositivo
  HANDLE_CUDA_ERROR(hipMemcpy(band1_d, band1, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH, hipMemcpyHostToDevice));
  HANDLE_CUDA_ERROR(hipMemcpy(band2_d, band2, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH, hipMemcpyHostToDevice));
  HANDLE_CUDA_ERROR(hipMemcpy(band3_d, band3, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH, hipMemcpyHostToDevice));
  HANDLE_CUDA_ERROR(hipMemcpy(band4_d, band4, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH, hipMemcpyHostToDevice));
  HANDLE_CUDA_ERROR(hipMemcpy(band5_d, band5, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH, hipMemcpyHostToDevice));
  HANDLE_CUDA_ERROR(hipMemcpy(band6_d, band6, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH, hipMemcpyHostToDevice));
  HANDLE_CUDA_ERROR(hipMemcpy(band7_d, band7, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH, hipMemcpyHostToDevice));
  HANDLE_CUDA_ERROR(hipMemcpy(band8_d, band8, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH, hipMemcpyHostToDevice));

  HANDLE_CUDA_ERROR(hipMemcpy(radiance1_d, radiance1, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH, hipMemcpyHostToDevice));
  HANDLE_CUDA_ERROR(hipMemcpy(radiance2_d, radiance2, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH, hipMemcpyHostToDevice));
  HANDLE_CUDA_ERROR(hipMemcpy(radiance3_d, radiance3, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH, hipMemcpyHostToDevice));
  HANDLE_CUDA_ERROR(hipMemcpy(radiance4_d, radiance4, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH, hipMemcpyHostToDevice));
  HANDLE_CUDA_ERROR(hipMemcpy(radiance5_d, radiance5, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH, hipMemcpyHostToDevice));
  HANDLE_CUDA_ERROR(hipMemcpy(radiance6_d, radiance6, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH, hipMemcpyHostToDevice));
  HANDLE_CUDA_ERROR(hipMemcpy(radiance7_d, radiance7, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH, hipMemcpyHostToDevice));
  HANDLE_CUDA_ERROR(hipMemcpy(radiance8_d, radiance8, sizeof(float) * MATRIX_HEIGTH * MATRIX_WIDTH, hipMemcpyHostToDevice));

  // Define o handle cutensor
  hiptensorHandle_t handle;
  HANDLE_ERROR(hiptensorCreate(&handle));

  // Define os descriptors
  hiptensorTensorDescriptor_t descA;
  hiptensorTensorDescriptor_t descB;
  hiptensorTensorDescriptor_t descC;
  HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle, &descA, tamanhoEixoA, dimensoesEixoA.data(), NULL, HIPTENSOR_R_32F, kAlignment));
  HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle, &descB, tamanhoEixoB, dimensoesEixoB.data(), NULL, HIPTENSOR_R_32F, kAlignment));
  HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle, &descC, tamanhoEixoC, dimensoesEixoC.data(), NULL, HIPTENSOR_R_32F, kAlignment));

  // Create Contraction Descriptor
  hiptensorOperationDescriptor_t desc;
  const hiptensorComputeDescriptor_t descCompute = HIPTENSOR_COMPUTE_DESC_32F;
  HANDLE_ERROR(hiptensorCreateContraction(handle,
                                         &desc,
                                         descA, indicesEixoA.data(), /* unary operator A*/ HIPTENSOR_OP_IDENTITY,
                                         descB, indicesEixoB.data(), /* unary operator B*/ HIPTENSOR_OP_IDENTITY,
                                         descC, indicesEixoC.data(), /* unary operator C*/ HIPTENSOR_OP_IDENTITY,
                                         descC, indicesEixoC.data(),
                                         descCompute));

  // Optional (but recommended): ensure that the scalar type is correct.
  hiptensorDataType_t scalarType;
  HANDLE_ERROR(hiptensorOperationDescriptorGetAttribute(handle, desc, HIPTENSOR_OPERATION_DESCRIPTOR_SCALAR_TYPE, (void *)&scalarType, sizeof(scalarType)));

  assert(scalarType == HIPTENSOR_R_32F);
  typedef float floatTypeCompute;
  floatTypeCompute alpha = (floatTypeCompute)1;
  floatTypeCompute beta = (floatTypeCompute)0;

  // Set the algorithm to use
  const hiptensorAlgo_t algo = HIPTENSOR_ALGO_DEFAULT;
  hiptensorPlanPreference_t planPref;
  HANDLE_ERROR(hiptensorCreatePlanPreference(handle, &planPref, algo, HIPTENSOR_JIT_MODE_NONE));

  // Query workspace estimate
  uint64_t workspaceSizeEstimate = 0;
  const hiptensorWorksizePreference_t workspacePref = HIPTENSOR_WORKSPACE_DEFAULT;
  HANDLE_ERROR(hiptensorEstimateWorkspaceSize(handle, desc, planPref, workspacePref, &workspaceSizeEstimate));

  // Create Contraction Plan
  hiptensorPlan_t plan;
  HANDLE_ERROR(hiptensorCreatePlan(handle, &plan, desc, planPref, workspaceSizeEstimate));

  // Optional: query actually used workspace
  uint64_t actualWorkspaceSize = 0;
  HANDLE_ERROR(hiptensorPlanGetAttribute(handle, plan, HIPTENSOR_PLAN_REQUIRED_WORKSPACE, &actualWorkspaceSize, sizeof(actualWorkspaceSize)));
  assert(actualWorkspaceSize <= workspaceSizeEstimate);

  // Define the workspace
  void *work = nullptr;
  if (actualWorkspaceSize > 0)
  {
    HANDLE_CUDA_ERROR(hipMalloc(&work, actualWorkspaceSize));
    assert(uintptr_t(work) % 128 == 0); // workspace must be aligned to 128 byte-boundary
  }

  // Execute
  hipStream_t stream;
  HANDLE_CUDA_ERROR(hipStreamCreate(&stream));

  begin = system_clock::now();

  // ==== Formula (contraction)
  // radiance_vector = 1 * bands_resampled * sensor.parameters + 1 * sensor.parameters[1][sensor.BRESCALE]

  HANDLE_ERROR(hiptensorContract(handle,
                                plan,
                                (void *)&alpha, band1_d, sensor.parameters[1][sensor.GRESCALE],
                                (void *)&beta, sensor.parameters[1][sensor.BRESCALE],
                                radiance1_d,
                                work,
                                actualWorkspaceSize,
                                stream));

  HANDLE_ERROR(hiptensorContract(handle,
                                plan,
                                (void *)&alpha, band2_d, sensor.parameters[2][sensor.GRESCALE],
                                (void *)&beta, sensor.parameters[2][sensor.BRESCALE],
                                radiance2_d,
                                work,
                                actualWorkspaceSize,
                                stream));

  HANDLE_ERROR(hiptensorContract(handle,
                                plan,
                                (void *)&alpha, band3_d, sensor.parameters[3][sensor.GRESCALE],
                                (void *)&beta, sensor.parameters[3][sensor.BRESCALE],
                                radiance3_d,
                                work,
                                actualWorkspaceSize,
                                stream));

  HANDLE_ERROR(hiptensorContract(handle,
                                plan,
                                (void *)&alpha, band4_d, sensor.parameters[4][sensor.GRESCALE],
                                (void *)&beta, sensor.parameters[4][sensor.BRESCALE],
                                radiance4_d,
                                work,
                                actualWorkspaceSize,
                                stream)); 

  HANDLE_ERROR(hiptensorContract(handle,
                                plan,
                                (void *)&alpha, band5_d, sensor.parameters[5][sensor.GRESCALE],
                                (void *)&beta, sensor.parameters[5][sensor.BRESCALE],
                                radiance5_d,
                                work,
                                actualWorkspaceSize,
                                stream));

  HANDLE_ERROR(hiptensorContract(handle,
                                plan,
                                (void *)&alpha, band6_d, sensor.parameters[6][sensor.GRESCALE],
                                (void *)&beta, sensor.parameters[6][sensor.BRESCALE],
                                radiance6_d,
                                work,
                                actualWorkspaceSize,
                                stream));

  HANDLE_ERROR(hiptensorContract(handle,
                                plan,
                                (void *)&alpha, band7_d, sensor.parameters[7][sensor.GRESCALE],
                                (void *)&beta, sensor.parameters[7][sensor.BRESCALE],
                                radiance7_d,
                                work,
                                actualWorkspaceSize,
                                stream));

  HANDLE_ERROR(hiptensorContract(handle,
                                plan,
                                (void *)&alpha, band8_d, sensor.parameters[8][sensor.GRESCALE],
                                (void *)&beta, sensor.parameters[8][sensor.BRESCALE],
                                radiance8_d,
                                work,
                                actualWorkspaceSize,
                                stream));

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end.time_since_epoch() - begin.time_since_epoch()).count();
  std::cout << "CUTENSOR," << MATRIX_HEIGTH << " x " << MATRIX_WIDTH << ", " << general_time << std::endl;
}

int main()
{
  TIFF *bands_resampled[8];

  // Sensor setup
  MTL mtl = MTL("./input/scenes/MTL.txt");
  Sensor sensor = Sensor(mtl.number_sensor, mtl.year);

  // TIFFs Setup
  std::string bands_paths[] = {
      "./input/scenes/B2.tif",
      "./input/scenes/B3.tif",
      "./input/scenes/B4.tif",
      "./input/scenes/B5.tif",
      "./input/scenes/B6.tif",
      "./input/scenes/B10.tif",
      "./input/scenes/B7.tif",
      "./input/scenes/final_tal.tif"};

  for (int i = 0; i < 8; i++)
  {
    std::string path_tiff_base = bands_paths[i];
    bands_resampled[i] = TIFFOpen(path_tiff_base.c_str(), "rm");
  }

  uint16_t sample_format;
  uint32_t height, width;
  TIFFGetField(bands_resampled[1], TIFFTAG_IMAGELENGTH, &height);
  TIFFGetField(bands_resampled[1], TIFFTAG_IMAGEWIDTH, &width);
  TIFFGetField(bands_resampled[1], TIFFTAG_SAMPLEFORMAT, &sample_format);

  for (int i = 0; i < 8; i++)
  {
    for (int line = 0; line < height; line++)
    {
      TIFF *curr_band = bands_resampled[i];
      tdata_t band_line_buff = _TIFFmalloc(TIFFScanlineSize(curr_band));
      unsigned short curr_band_line_size = TIFFScanlineSize(curr_band) / width;
      TIFFReadScanline(curr_band, band_line_buff, line);

      for (int col = 0; col < width; col++)
      {
        float value = 0;
        memcpy(&value, static_cast<unsigned char *>(band_line_buff) + col * curr_band_line_size, curr_band_line_size);

        switch (i)
        {
        case 0:
          band1[line * width + col] = value;
          break;
        case 1:
          band2[line * width + col] = value;
          break;
        case 2:
          band3[line * width + col] = value;
          break;
        case 3:
          band4[line * width + col] = value;
          break;
        case 4:
          band5[line * width + col] = value;
          break;
        case 5:
          band6[line * width + col] = value;
          break;
        case 6:
          band7[line * width + col] = value;
          break;
        case 7:
          band8[line * width + col] = value;
          break;
        default:
            break;  
        }
      }
      _TIFFfree(band_line_buff);
    }
  }

  // ======== RUN =========
  serial(sensor, mtl);
  cutensor(sensor, mtl);
  // ======== RUN =========

  for (int i = 0; i < 8; i++)
  {
    TIFFClose(bands_resampled[i]);
  }

  return 0;
}