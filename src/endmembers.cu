#include "hip/hip_runtime.h"
#include "endmembers.h"

void compute_H0(vector<double> net_radiation_line, vector<double> soil_heat_flux, int width_band, vector<double> &ho_line)
{
  for (int col = 0; col < width_band; col++)
    ho_line[col] = net_radiation_line[col] - soil_heat_flux[col];
};

void filter_valid_values(vector<double> target_line, double *target_values, int width_band, int *pos)
{
  for (int col = 0; col < width_band; col++)
  {
    if (!isnan(target_line[col]) && !isinf(target_line[col]))
    {
      target_values[*pos] = target_line[col];
      (*pos)++;
    }
  }
}

void get_quartiles(vector<vector<double>> target_vector, double *v_quartile, int height_band, int width_band, double first_interval, double middle_interval, double last_interval)
{
  const int SIZE = height_band * width_band;
  double *target_values = (double *)malloc(sizeof(double) * SIZE);

  if (target_values == NULL)
    exit(15);

  int pos = 0;
  for (int line = 0; line < height_band; line++)
  {
    filter_valid_values(target_vector[line], target_values, width_band, &pos);
  }

  int first_index = static_cast<int>(floor(first_interval * pos));
  int middle_index = static_cast<int>(floor(middle_interval * pos));
  int last_index = static_cast<int>(floor(last_interval * pos));

  std::nth_element(target_values, target_values + first_index, target_values + pos);
  v_quartile[0] = target_values[first_index];

  std::nth_element(target_values, target_values + middle_index, target_values + pos);
  v_quartile[1] = target_values[middle_index];

  std::nth_element(target_values, target_values + last_index, target_values + pos);
  v_quartile[2] = target_values[last_index];

  free(target_values);
}


pair<Candidate, Candidate> getEndmembersASEBAL(vector<vector<double>> ndvi_vector, vector<vector<double>> surface_temperature_vector, vector<vector<double>> albedo_vector, vector<vector<double>> net_radiation_vector, vector<vector<double>> soil_heat_vector, int height_band, int width_band)
{
  vector<Candidate> hotCandidates;
  vector<Candidate> coldCandidates;
  vector<vector<double>> ho_vector(height_band, vector<double>(width_band));

  vector<double> tsQuartile(3);
  vector<double> ndviQuartile(3);
  vector<double> albedoQuartile(3);
  get_quartiles(ndvi_vector, ndviQuartile.data(), height_band, width_band, 0.25, 0.75, 0.75);
  get_quartiles(albedo_vector, albedoQuartile.data(), height_band, width_band, 0.25, 0.50, 0.75);
  get_quartiles(surface_temperature_vector, tsQuartile.data(), height_band, width_band, 0.25, 0.75, 0.75);

  for (int line = 0; line < height_band; line++)
  {
    vector<double> ho_line = ho_vector[line];
    vector<double> ndvi_line = ndvi_vector[line];
    vector<double> surface_temperature_line = surface_temperature_vector[line];
    vector<double> albedo_line = albedo_vector[line];
    vector<double> net_radiation_line = net_radiation_vector[line];
    vector<double> soil_heat_line = soil_heat_vector[line];

    compute_H0(net_radiation_line, soil_heat_line, width_band, ho_line);

    for (int col = 0; col < width_band; col++)
    {

      bool hotAlbedo = !isnan(albedo_line[col]) && albedo_line[col] > albedoQuartile[1];
      bool hotNDVI = !isnan(ndvi_line[col]) && ndvi_line[col] > 0.10 && ndvi_line[col] < ndviQuartile[0];
      bool hotTS = !isnan(surface_temperature_line[col]) && surface_temperature_line[col] > tsQuartile[1];

      bool coldAlbedo = !isnan(albedo_line[col]) && albedo_line[col] < albedoQuartile[1];
      bool coldNDVI = !isnan(ndvi_line[col]) && ndvi_line[col] >= ndviQuartile[1];
      bool coldTS = !isnan(surface_temperature_line[col]) && surface_temperature_line[col] < tsQuartile[0];

      if (hotAlbedo && hotNDVI && hotTS)
        hotCandidates.emplace_back(ndvi_line[col], surface_temperature_line[col], net_radiation_line[col], soil_heat_line[col], ho_line[col], line, col);

      if (coldAlbedo && coldNDVI && coldTS)
        coldCandidates.emplace_back(ndvi_line[col], surface_temperature_line[col], net_radiation_line[col], soil_heat_line[col], ho_line[col], line, col);
    }
  }

  if (hotCandidates.empty() || coldCandidates.empty())
  {
    cerr << "Pixel problem! - There are no final candidates";
    exit(15);
  }

  // Creating second pixel group, all values lower than the 3rd quartile are excluded
  std::sort(hotCandidates.begin(), hotCandidates.end(), compare_candidate_temperature);
  std::sort(coldCandidates.begin(), coldCandidates.end(), compare_candidate_temperature);
  
  unsigned int hotPos = static_cast<unsigned int>(std::floor(hotCandidates.size() * 0.5));
  unsigned int coldPos = static_cast<unsigned int>(std::floor(coldCandidates.size() * 0.5));

  return {hotCandidates[hotPos], coldCandidates[coldPos]};
}

pair<Candidate, Candidate> getEndmembersSTEPP(vector<vector<double>> ndvi_vector, vector<vector<double>> surface_temperature_vector, vector<vector<double>> albedo_vector, vector<vector<double>> net_radiation_vector, vector<vector<double>> soil_heat_vector, int height_band, int width_band)
{
  vector<Candidate> hotCandidates;
  vector<Candidate> coldCandidates;

  vector<vector<double>> ho_vector(height_band, vector<double>(width_band));

  vector<double> tsQuartile(3);
  vector<double> ndviQuartile(3);
  vector<double> albedoQuartile(3);
  get_quartiles(ndvi_vector, ndviQuartile.data(), height_band, width_band, 0.15, 0.85, 0.97);
  get_quartiles(albedo_vector, albedoQuartile.data(), height_band, width_band, 0.25, 0.50, 0.75);
  get_quartiles(surface_temperature_vector, tsQuartile.data(), height_band, width_band, 0.20, 0.85, 0.97);

  for (int line = 0; line < height_band; line++)
  {
    vector<double> ho_line = ho_vector[line];
    vector<double> ndvi_line = ndvi_vector[line];
    vector<double> surface_temperature_line = surface_temperature_vector[line];
    vector<double> albedo_line = albedo_vector[line];
    vector<double> net_radiation_line = net_radiation_vector[line];
    vector<double> soil_heat_line = soil_heat_vector[line];

    compute_H0(net_radiation_line, soil_heat_line, width_band, ho_line);

    for (int col = 0; col < width_band; col++)
    {
      bool hotNDVI = !std::isnan(ndvi_line[col]) && ndvi_line[col] > 0.10 && ndvi_line[col] < ndviQuartile[0];
      bool hotAlbedo = !std::isnan(albedo_line[col]) && albedo_line[col] > albedoQuartile[1] && albedo_line[col] < albedoQuartile[2];
      bool hotTS = !std::isnan(surface_temperature_line[col]) && surface_temperature_line[col] > tsQuartile[1] && surface_temperature_line[col] < tsQuartile[2];

      bool coldNDVI = !std::isnan(ndvi_line[col]) && ndvi_line[col] > ndviQuartile[2];
      bool coldAlbedo = !std::isnan(surface_temperature_line[col]) && albedo_line[col] > albedoQuartile[0] && albedo_line[col] < albedoQuartile[1];
      bool coldTS = !std::isnan(albedo_line[col]) && surface_temperature_line[col] < tsQuartile[0];

      if (hotAlbedo && hotNDVI && hotTS)
        hotCandidates.emplace_back(ndvi_line[col], surface_temperature_line[col], net_radiation_line[col], soil_heat_line[col], ho_line[col], line, col);

      if (coldNDVI && coldAlbedo && coldTS)
        coldCandidates.emplace_back(ndvi_line[col], surface_temperature_line[col], net_radiation_line[col], soil_heat_line[col], ho_line[col], line, col);
    }
  }

  if (hotCandidates.empty() || coldCandidates.empty())
  {
    cerr << "Pixel problem! - There are no final candidates";
    exit(15);
  }

  std::sort(hotCandidates.begin(), hotCandidates.end(), compare_candidate_temperature);
  std::sort(coldCandidates.begin(), coldCandidates.end(), compare_candidate_temperature);

  unsigned int hotPos = static_cast<unsigned int>(std::floor(hotCandidates.size() * 0.5));
  unsigned int coldPos = static_cast<unsigned int>(std::floor(coldCandidates.size() * 0.5));

  return {hotCandidates[hotPos], coldCandidates[coldPos]};
}
