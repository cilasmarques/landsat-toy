#include "hip/hip_runtime.h"
#include "kernels.cuh"

__global__ void rah_correction_cycle_STEEP(double *surface_temperature_pointer, double *d0_pointer, double *kb1_pointer, double *zom_pointer, double *ustarR_pointer,
                                           double *ustarW_pointer, double *rahR_pointer, double *rahW_pointer, double *H_pointer, double a, double b, int height,
                                           int width)
{
  // Identify position
  unsigned int col = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int row = threadIdx.y + blockIdx.y * blockDim.y;

  if (col < width && row < height)
  {
    unsigned int pos = row * width + col;

    double DISP = d0_pointer[pos];
    double dT_ini_terra = a + b * (surface_temperature_pointer[pos] - 273.15);

    double sensibleHeatFlux = RHO * SPECIFIC_HEAT_AIR * (dT_ini_terra) / rahR_pointer[pos];
    double L = -1 * ((RHO * SPECIFIC_HEAT_AIR * pow(ustarR_pointer[pos], 3) * surface_temperature_pointer[pos]) / (VON_KARMAN * GRAVITY * sensibleHeatFlux));

    double y2 = pow((1 - (16 * (10 - DISP)) / L), 0.25);
    double x200 = pow((1 - (16 * (10 - DISP)) / L), 0.25);

    double psi2, psi200;
    if (!isnan(L) && L > 0)
    {
      psi2 = -5 * ((10 - DISP) / L);
      psi200 = -5 * ((10 - DISP) / L);
    }
    else
    {
      psi2 = 2 * log((1 + y2 * y2) / 2);
      psi200 = 2 * log((1 + x200) / 2) + log((1 + x200 * x200) / 2) - 2 * atan(x200) + 0.5 * M_PI;
    }

    double ust = (VON_KARMAN * ustarR_pointer[pos]) / (log((10 - DISP) / zom_pointer[pos]) - psi200);

    double zoh_terra = zom_pointer[pos] / pow(exp(1.0), (kb1_pointer[pos]));
    double temp_rah1_corr_terra = (ust * VON_KARMAN);
    double temp_rah2_corr_terra = log((10 - DISP) / zom_pointer[pos]) - psi2;
    double temp_rah3_corr_terra = temp_rah1_corr_terra * log(zom_pointer[pos] / zoh_terra);
    double rah = (temp_rah1_corr_terra * temp_rah2_corr_terra) + temp_rah3_corr_terra;

    ustarW_pointer[pos] = ust;
    rahW_pointer[pos] = rah;
    H_pointer[pos] = sensibleHeatFlux;
  }
}