#include "hip/hip_runtime.h"
#include <iostream>
#include <hiptensor.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <vector>

using namespace std::chrono;

// Define the matrix size
#define MATRIX_HEIGTH 4
#define MATRIX_WIDTH 4

__global__ void matrixMn(float *d_matrixA, float *d_matrixB, float *d_matrixC)
{
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int row = idx / MATRIX_WIDTH;
  unsigned int col = idx % MATRIX_WIDTH;

  if (row < MATRIX_HEIGTH && col < MATRIX_WIDTH)
  {
    //// Produto Escalar
    // for (int i = 0; i < MATRIX_WIDTH; i++)
    // {
    //   d_matrixC[row * MATRIX_WIDTH + col] += d_matrixA[row * MATRIX_WIDTH + i] * d_matrixB[i * MATRIX_WIDTH + col];
    // }

    ///// Produto de hadamard
    unsigned int pos = row * MATRIX_WIDTH + col;
    d_matrixC[pos] = d_matrixA[pos] * d_matrixB[pos];
  }
}

void cudaCoreExecution(float *d_matrixA, float *d_matrixB, float *d_matrixC)
{
  // Perform matrix sum: C = A * B
  int num_threads = 4;
  int num_blocks = ceil(MATRIX_HEIGTH * MATRIX_WIDTH / num_threads);
  matrixMn<<<num_blocks, num_threads>>>(d_matrixA, d_matrixB, d_matrixC);
}

int main()
{
  system_clock::time_point begin, end;
  int64_t general_time;

  // Create a matrix
  float *matrixA = new float[MATRIX_HEIGTH * MATRIX_WIDTH];
  float *matrixB = new float[MATRIX_HEIGTH * MATRIX_WIDTH];

  // Initialize the matrix
  for (int i = 0; i < MATRIX_HEIGTH * MATRIX_WIDTH; i++)
  {
    matrixA[i] = 4.0f;
    matrixB[i] = 2.0f;
  }

  // Allocate the memory on the device
  float *d_matrixA;
  float *d_matrixB;
  float *d_matrixC;
  hipMalloc(&d_matrixA, MATRIX_HEIGTH * MATRIX_WIDTH * sizeof(float));
  hipMalloc(&d_matrixB, MATRIX_HEIGTH * MATRIX_WIDTH * sizeof(float));
  hipMalloc(&d_matrixC, MATRIX_HEIGTH * MATRIX_WIDTH * sizeof(float));

  // Copy the data to the device
  hipMemcpy(d_matrixA, matrixA, MATRIX_HEIGTH * MATRIX_WIDTH * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_matrixB, matrixB, MATRIX_HEIGTH * MATRIX_WIDTH * sizeof(float), hipMemcpyHostToDevice);

  // ======== RUN CUDA CORE =========
  begin = system_clock::now();

  cudaCoreExecution(d_matrixA, d_matrixB, d_matrixC);
  hipDeviceSynchronize();

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end.time_since_epoch() - begin.time_since_epoch()).count();
  std::cout << "CUDA CORE - TOTAL TIME (ns): " << general_time << std::endl;
  // ======== RUN CUDA CORE =========

  // Copy the result back to the host
  float *matrixC = new float[MATRIX_HEIGTH * MATRIX_WIDTH];
  hipMemcpy(matrixC, d_matrixC, MATRIX_HEIGTH * MATRIX_WIDTH * sizeof(float), hipMemcpyDeviceToHost);

  // Print the result
  // std::cout << "Matrix C: " << std::endl;
  // for (int i = 0; i < MATRIX_HEIGTH; i++)
  // {
  //   for (int j = 0; j < MATRIX_WIDTH; j++)
  //   {
  //     std::cout << matrixC[i * MATRIX_WIDTH + j] << " ";
  //   }
  //   std::cout << std::endl;
  // }

  // Free the memory
  delete[] matrixA;
  delete[] matrixB;
  delete[] matrixC;
  hipFree(d_matrixA);
  hipFree(d_matrixB);
  hipFree(d_matrixC);

  return 0;
}